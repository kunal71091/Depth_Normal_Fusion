#include "hip/hip_runtime.h"
#include <d3d_cudaBase/cudaCommon.h>

namespace D3D_CUDA
{
namespace CudaDepthIntegratorIROS2011DeviceCode
{


void initTile(int xRes, int yRes, int zRes, float** voxelsD)
{
    const int numBytes = xRes*yRes*zRes*sizeof(float);
    D3D_CUDA_CHECKED_CALL(hipMalloc(voxelsD, numBytes);)
    D3D_CUDA_CHECKED_CALL(hipMemset(*voxelsD, 0, numBytes);)
}

void releaseDeviceMemory(void *addr)
{
    D3D_CUDA_CHECKED_CALL(hipFree(addr);)
}

void allocateAndUploadTransformation(float* transformationH, float** transformationD)
{
    D3D_CUDA_CHECKED_CALL(hipMalloc(transformationD, 4*4*sizeof(float));)
    D3D_CUDA_CHECKED_CALL(hipMemcpy(*transformationD, transformationH, 4*4*sizeof(float), hipMemcpyHostToDevice);)
}

void allocateAndUploadProjection(float* projectionH, float** projectionD)
{
    D3D_CUDA_CHECKED_CALL(hipMalloc(projectionD, 3*4*sizeof(float));)
    D3D_CUDA_CHECKED_CALL(hipMemcpy(*projectionD, projectionH, 3*4*sizeof(float), hipMemcpyHostToDevice);)
}

void allocateAndUploadDepthData(float* depthDataH, int rows, int cols, float** depthDataD)
{
    D3D_CUDA_CHECKED_CALL(hipMalloc(depthDataD, rows*cols*sizeof(float));)
    D3D_CUDA_CHECKED_CALL(hipMemcpy(*depthDataD, depthDataH, rows*cols*sizeof(float), hipMemcpyHostToDevice);)
}

void downloadTile(int xRes, int yRes, int zRes, float* voxelsD, float* voxelsH)
{
    const int numBytes = xRes*yRes*zRes*sizeof(float);
    D3D_CUDA_CHECKED_CALL(hipMemcpy(voxelsH, voxelsD, numBytes, hipMemcpyDeviceToHost);)
}

__device__ void project(float* projMat, float* point, float* projPoint)
{
    projPoint[0] = projMat[0]*point[0] + projMat[1]*point[1] + projMat[2]*point[2] + projMat[3];
    projPoint[1] = projMat[4]*point[0] + projMat[5]*point[1] + projMat[6]*point[2] + projMat[7];
    projPoint[2] = projMat[8]*point[0] + projMat[9]*point[1] + projMat[10]*point[2] + projMat[11];
}

__device__ void transform(float* transformMat, float* point, float* transfPoint)
{
    transfPoint[0] = transformMat[0]*point[0] + transformMat[1]*point[1] + transformMat[2]*point[2] + transformMat[3];
    transfPoint[1] = transformMat[4]*point[0] + transformMat[5]*point[1] + transformMat[6]*point[2] + transformMat[7];
    transfPoint[2] = transformMat[8]*point[0] + transformMat[9]*point[1] + transformMat[10]*point[2] + transformMat[11];
}

__global__ void integrationKernel(float* voxelD, float* boxToGlobalD,
                             int xRes, int yRes, int zRes,
                             float xBegin, float yBegin, float zBegin,
                             float deltaX, float deltaY, float deltaZ,
                             float* depthDataD, int rows, int cols,
                             float maxDepth, float minDepth,
                             float epsilon, float eta, float uncertFact,
                             float* projD)
{
    __shared__ float proj[12];
    __shared__ float boxToGlobal[16];

    if (threadIdx.x < 12)
    {
        proj[threadIdx.x] = projD[threadIdx.x];
    }
    if (threadIdx.x < 16)
    {
        boxToGlobal[threadIdx.x] = boxToGlobalD[threadIdx.x];
    }
    __syncthreads();

    const int x = blockIdx.x;
    const int y = blockIdx.y;
    const int z = threadIdx.x;

    // voxel center in box coordinate frame
    float voxCenterPoint[3];
    voxCenterPoint[0] = xBegin + x*deltaX;
    voxCenterPoint[1] = yBegin + y*deltaY;
    voxCenterPoint[2] = zBegin + z*deltaZ;

    float voxCenterPointG[3];

    transform(boxToGlobal, voxCenterPoint, voxCenterPointG);

    float voxCenterProj[3];
    project(proj, voxCenterPointG, voxCenterProj);

    if (voxCenterProj[2] >= minDepth && voxCenterProj[2] <= maxDepth)
    {

        // perspective division
        int xp,yp;
        xp = round(voxCenterProj[0]/voxCenterProj[2]);
        yp = round(voxCenterProj[1]/voxCenterProj[2]);

        // test if inside image
        if (xp >= 0 && xp < cols && yp >= 0 && yp < rows) {

            float depth = depthDataD[yp*cols + xp];

            if (depth > 0)
            {

                float dist = voxCenterProj[2] - depth;
                float distToNextDisparityPlane = max(((depth*depth)*uncertFact), epsilon);

                if (dist < distToNextDisparityPlane)
                {

                    // debug
//                    int voxInd = z*xRes*yRes + y*xRes + x;
//                    voxelD[voxInd] += 1;

                    const int voxInd = z*xRes*yRes + y*xRes + x;

                    // inside band
                    if (fabs(dist) < distToNextDisparityPlane) {
                        if (dist < 0) {
                            // in front of surface
                            voxelD[voxInd] -= epsilon/distToNextDisparityPlane;
                        } else {
                            // behind surface
                            voxelD[voxInd] += epsilon/distToNextDisparityPlane;
                        }
                        // viewing ray
                    } else {
                        voxelD[voxInd] -= eta*epsilon/distToNextDisparityPlane;
                    }

                }
            }
        }
    }



}

void integrateDepthMap(float* voxelD, float* boxToGlobalD,
                       int xRes, int yRes, int zRes,
                       float xBegin, float yBegin, float zBegin,
                       float deltaX, float deltaY, float deltaZ,
                       float* depthDataD, int rows, int cols,
                       float maxDepth, float minDepth,
                       float epsilon, float eta, float uncertFact,
                       float* projD)
{
    dim3 dimGrid(xRes, yRes);
    dim3 dimBlock(zRes);

    integrationKernel<<<dimGrid,dimBlock>>>(voxelD, boxToGlobalD,
                                            xRes, yRes, zRes,
                                            xBegin, yBegin, zBegin,
                                            deltaX, deltaY, deltaZ,
                                            depthDataD, rows, cols,
                                            maxDepth, minDepth,
                                            epsilon, eta, uncertFact,
                                            projD);
    D3D_CUDA_CHECK_ERROR

}

}
}

