#include "hip/hip_runtime.h"
#include <d3d_cudaBase/cudaCommon.h>
#include <iostream>

namespace D3D_CUDA
{
namespace CudaDepthIntegratorMultiClassv2DeviceCode
{


void initTile(int xRes, int yRes, int zRes, int numClasses, float** voxelsD)
{
    const int numBytes = xRes*yRes*zRes*numClasses*sizeof(float);
    D3D_CUDA_CHECKED_CALL(hipMalloc(voxelsD, numBytes);)
    D3D_CUDA_CHECKED_CALL(hipMemset(*voxelsD, 0, numBytes);)
}

void releaseDeviceMemory(void *addr)
{
    D3D_CUDA_CHECKED_CALL(hipFree(addr);)
}

void allocateAndUploadTransformation(float* transformationH, float** transformationD)
{
    D3D_CUDA_CHECKED_CALL(hipMalloc(transformationD, 4*4*sizeof(float));)
    D3D_CUDA_CHECKED_CALL(hipMemcpy(*transformationD, transformationH, 4*4*sizeof(float), hipMemcpyHostToDevice);)
}

void allocateAndUploadProjection(float* projectionH, float** projectionD)
{
    D3D_CUDA_CHECKED_CALL(hipMalloc(projectionD, 3*4*sizeof(float));)
    D3D_CUDA_CHECKED_CALL(hipMemcpy(*projectionD, projectionH, 3*4*sizeof(float), hipMemcpyHostToDevice);)
}

void allocateAndUploadDepthData(float* depthDataH, int rows, int cols, float** depthDataD)
{
    D3D_CUDA_CHECKED_CALL(hipMalloc(depthDataD, rows*cols*sizeof(float));)
    D3D_CUDA_CHECKED_CALL(hipMemcpy(*depthDataD, depthDataH, rows*cols*sizeof(float), hipMemcpyHostToDevice);)
}

void allocateAndUploadClassScoresData(float* classScoresDataH, int rows, int cols, int numClasses, float** classScoresDataD)
{
    D3D_CUDA_CHECKED_CALL(hipMalloc(classScoresDataD, rows*cols*numClasses*sizeof(float)); )
    D3D_CUDA_CHECKED_CALL(hipMemcpy(*classScoresDataD, classScoresDataH, rows*cols*numClasses*sizeof(float), hipMemcpyHostToDevice);)

}


void downloadTile(int xRes, int yRes, int zRes, float* voxelsD, float* voxelsH)
{
    const int numBytes = xRes*yRes*zRes*sizeof(float);
    D3D_CUDA_CHECKED_CALL(hipMemcpy(voxelsH, voxelsD, numBytes, hipMemcpyDeviceToHost);)
}

__device__ void project(float* projMat, float* point, float* projPoint)
{
    projPoint[0] = projMat[0]*point[0] + projMat[1]*point[1] + projMat[2]*point[2] + projMat[3];
    projPoint[1] = projMat[4]*point[0] + projMat[5]*point[1] + projMat[6]*point[2] + projMat[7];
    projPoint[2] = projMat[8]*point[0] + projMat[9]*point[1] + projMat[10]*point[2] + projMat[11];
}

__device__ void transform(float* transformMat, float* point, float* transfPoint)
{
    transfPoint[0] = transformMat[0]*point[0] + transformMat[1]*point[1] + transformMat[2]*point[2] + transformMat[3];
    transfPoint[1] = transformMat[4]*point[0] + transformMat[5]*point[1] + transformMat[6]*point[2] + transformMat[7];
    transfPoint[2] = transformMat[8]*point[0] + transformMat[9]*point[1] + transformMat[10]*point[2] + transformMat[11];
}

__global__ void integrationKernel(float* voxelD, const float* boxToGlobalD,
                             int xRes, int yRes, int zRes,
                             int numClasses, int freeSpaceClass,
                             float xBegin, float yBegin, float zBegin,
                             float deltaX, float deltaY, float deltaZ,
                             const float* classScoresDataD, const float* depthDataD, int rows, int cols,
                             float maxDepth, float minDepth,
                             float epsilon, float eta, float uncertFact, float rho,
                             const float* projD)
{
    __shared__ float proj[12];
    __shared__ float boxToGlobal[16];

    if (threadIdx.x < 12)
    {
        proj[threadIdx.x] = projD[threadIdx.x];
    }
    if (threadIdx.x < 16)
    {
        boxToGlobal[threadIdx.x] = boxToGlobalD[threadIdx.x];
    }
    __syncthreads();

    const int x = blockIdx.x;
    const int y = blockIdx.y;
    const int z = threadIdx.x;

    // voxel center in box coordinate frame
    float voxCenterPoint[3];
    voxCenterPoint[0] = xBegin + x*deltaX;
    voxCenterPoint[1] = yBegin + y*deltaY;
    voxCenterPoint[2] = zBegin + z*deltaZ;

    const float voxelDiameter = sqrtf(deltaX*deltaX + deltaY*deltaY + deltaZ*deltaZ);

    float voxCenterPointG[3];

    transform(boxToGlobal, voxCenterPoint, voxCenterPointG);

    float voxCenterProj[3];
    project(proj, voxCenterPointG, voxCenterProj);

    if (voxCenterProj[2] >= minDepth && voxCenterProj[2] <= maxDepth)
    {

        // perspective division
        int xp,yp;
        xp = round(voxCenterProj[0]/voxCenterProj[2]);
        yp = round(voxCenterProj[1]/voxCenterProj[2]);

        // test if inside image
        if (xp >= 0 && xp < cols && yp >= 0 && yp < rows)
        {

            const int depthMapIdx = yp*cols + xp;

            float depth = depthDataD[depthMapIdx];

// all transition approach

            int voxInd = (z*xRes*yRes + y*xRes + x)*numClasses;

            float bestScore = classScoresDataD[depthMapIdx*numClasses];
            int bestClass = 0;

            for (int i = 1; i < numClasses; i++)
            {
                if (classScoresDataD[depthMapIdx*numClasses + i] > bestScore)
                {
                    bestScore = classScoresDataD[depthMapIdx*numClasses + i];
                    bestClass = i;
                }
            }

            if (bestClass == freeSpaceClass)
            {
                voxelD[voxInd + freeSpaceClass] -= eta;
            }
            else
            {
                if (depth > 0)
                {

                    float dist = voxCenterProj[2] - depth;

                    // inside band
                    if (fabs(dist) < epsilon)
                    {
                        if (dist < 0)
                        {
                            // in front of surface
                            voxelD[voxInd + freeSpaceClass] -= 1;

                        }
                        else
                        {
                            // behind surface
                            voxelD[voxInd + bestClass] -= 1;
                        }
                    }
                }
            }
        }
    }



}

void integrateDepthMap(float* voxelD, const float* boxToGlobalD,
                       int xRes, int yRes, int zRes,
                       int numClasses, int freeSpaceClass,
                       float xBegin, float yBegin, float zBegin,
                       float deltaX, float deltaY, float deltaZ,
                       const float* classScoresDataD, const float* depthDataD, int rows, int cols,
                       float maxDepth, float minDepth,
                       float epsilon, float eta, float uncertFact, float rho,
                       const float* projD)
{
    dim3 dimGrid(xRes, yRes);
    dim3 dimBlock(zRes);

    std::cout << "Eta = " << eta << std::endl;

    integrationKernel<<<dimGrid,dimBlock>>>(voxelD, boxToGlobalD,
                                            xRes, yRes, zRes,
                                            numClasses, freeSpaceClass,
                                            xBegin, yBegin, zBegin,
                                            deltaX, deltaY, deltaZ,
                                            classScoresDataD, depthDataD, rows, cols,
                                            maxDepth, minDepth,
                                            epsilon, eta, uncertFact, rho,
                                            projD);
    D3D_CUDA_CHECK_ERROR

}

}
}

