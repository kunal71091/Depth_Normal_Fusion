#include "hip/hip_runtime.h"
#include <d3d_cudaBase/cudaCommon.h>
#include <iostream>

namespace D3D_CUDA
{
namespace CudaDepthIntegratorDeviceCode
{


void initTile(int xRes, int yRes, int zRes, float** voxelsD)
{
    const int numBytes = xRes*yRes*zRes*sizeof(float);
    D3D_CUDA_CHECKED_CALL(hipMalloc(voxelsD, numBytes);)
    D3D_CUDA_CHECKED_CALL(hipMemset(*voxelsD, 0, numBytes);)
}

void releaseDeviceMemory(void *addr)
{
    D3D_CUDA_CHECKED_CALL(hipFree(addr);)
}

void allocateAndUploadTransformation(float* transformationH, float** transformationD)
{
    D3D_CUDA_CHECKED_CALL(hipMalloc(transformationD, 4*4*sizeof(float));)
    D3D_CUDA_CHECKED_CALL(hipMemcpy(*transformationD, transformationH, 4*4*sizeof(float), hipMemcpyHostToDevice);)
}

void allocateAndUploadProjection(float* projectionH, float** projectionD)
{
    D3D_CUDA_CHECKED_CALL(hipMalloc(projectionD, 3*4*sizeof(float));)
    D3D_CUDA_CHECKED_CALL(hipMemcpy(*projectionD, projectionH, 3*4*sizeof(float), hipMemcpyHostToDevice);)
}

void allocateAndUploadDepthData(float* depthDataH, int rows, int cols, float** depthDataD)
{
    D3D_CUDA_CHECKED_CALL(hipMalloc(depthDataD, rows*cols*sizeof(float));)
    D3D_CUDA_CHECKED_CALL(hipMemcpy(*depthDataD, depthDataH, rows*cols*sizeof(float), hipMemcpyHostToDevice);)
}

void downloadTile(int xRes, int yRes, int zRes, float* voxelsD, float* voxelsH)
{
    const int numBytes = xRes*yRes*zRes*sizeof(float);
    D3D_CUDA_CHECKED_CALL(hipMemcpy(voxelsH, voxelsD, numBytes, hipMemcpyDeviceToHost);)
}

__device__ void project(float* projMat, float* point, float* projPoint)
{
    projPoint[0] = projMat[0]*point[0] + projMat[1]*point[1] + projMat[2]*point[2] + projMat[3];
    projPoint[1] = projMat[4]*point[0] + projMat[5]*point[1] + projMat[6]*point[2] + projMat[7];
    projPoint[2] = projMat[8]*point[0] + projMat[9]*point[1] + projMat[10]*point[2] + projMat[11];
}

__device__ void transform(float* transformMat, float* point, float* transfPoint)
{
    transfPoint[0] = transformMat[0]*point[0] + transformMat[1]*point[1] + transformMat[2]*point[2] + transformMat[3];
    transfPoint[1] = transformMat[4]*point[0] + transformMat[5]*point[1] + transformMat[6]*point[2] + transformMat[7];
    transfPoint[2] = transformMat[8]*point[0] + transformMat[9]*point[1] + transformMat[10]*point[2] + transformMat[11];
}

__global__ void integrationKernelWithFixedWeights(float* voxelD, float* boxToGlobalD,
                                                  int xRes, int yRes, int zRes,
                                                  float xBegin, float yBegin, float zBegin,
                                                  float deltaX, float deltaY, float deltaZ,
                                                  float* depthDataD, int rows, int cols,
                                                  float maxDepth, float minDepth,
                                                  float epsilon, float eta, float delta1, float delta2, float delta3,
                                                  float* projD)
{
    __shared__ float proj[12];
    __shared__ float boxToGlobal[16];

    if (threadIdx.x < 12)
    {
        proj[threadIdx.x] = projD[threadIdx.x];
    }
    if (threadIdx.x < 16)
    {
        boxToGlobal[threadIdx.x] = boxToGlobalD[threadIdx.x];
    }
    __syncthreads();

    const int x = blockIdx.x;
    const int y = blockIdx.y;
    const int z = threadIdx.x;

    // voxel center in box coordinate frame
    float voxCenterPoint[3];
    voxCenterPoint[0] = xBegin + x*deltaX;
    voxCenterPoint[1] = yBegin + y*deltaY;
    voxCenterPoint[2] = zBegin + z*deltaZ;

    float voxCenterPointG[3];

    transform(boxToGlobal, voxCenterPoint, voxCenterPointG);

    float voxCenterProj[3];
    project(proj, voxCenterPointG, voxCenterProj);

    if (voxCenterProj[2] >= minDepth && voxCenterProj[2] <= maxDepth)
    {

        // perspective division
        int xp,yp;
        xp = round(voxCenterProj[0]/voxCenterProj[2]);
        yp = round(voxCenterProj[1]/voxCenterProj[2]);

        // test if inside image
        if (xp >= 0 && xp < cols && yp >= 0 && yp < rows) {

            float depth = depthDataD[yp*cols + xp];

            if (depth > 0)
            {
                float dist = voxCenterProj[2] - depth;

                if (fabs(dist) > delta1)
                {
                    const int voxInd = z*xRes*yRes + y*xRes + x;
                    if (fabs(dist) < delta2)
                    {
                        float weight = ((fabs(dist) - delta1)/(delta2 - delta1))*epsilon;
                        if (dist < 0)
                        {
                            // in front of surface
                            voxelD[voxInd] -= weight;
                        }
                        else if (dist > 0)
                        {
                            // behind surface
                            voxelD[voxInd] += weight;
                        }

                    }
                    else if(fabs(dist) < delta3)
                    {

                        if (dist < 0)
                        {
                            // in front of surface
                            voxelD[voxInd] -= epsilon;
                        }
                        else if (dist > 0)
                        {
                            // behind surface
                            voxelD[voxInd] += epsilon;
                        }
                    }
                    else
                    {
                        if (dist < 0)
                        {
                            voxelD[voxInd] -= eta*epsilon;
                        }
                    }

                }
            }
        }
    }
}


void initTileDebug(int xRes, int yRes, int zRes, float** voxelsD)
{
    const int numBytes = xRes*yRes*zRes*sizeof(float);
    *voxelsD = new float[xRes*yRes*zRes];
    memset(*voxelsD, 0, numBytes);
}

void releaseDeviceMemoryDebug(void *addr)
{
    delete [] addr;
}

void allocateAndUploadTransformationDebug(float* transformationH, float** transformationD)
{
    *transformationD = new float[4*4];
    memcpy(*transformationD, transformationH, 4*4*sizeof(float));
}

void allocateAndUploadProjectionDebug(float* projectionH, float** projectionD)
{
    *projectionD = new float[3*4];
    memcpy(*projectionD, projectionH, 3*4*sizeof(float));
}

void allocateAndUploadDepthDataDebug(float* depthDataH, int rows, int cols, float** depthDataD)
{
    *depthDataD = new float[rows*cols];
    memcpy(*depthDataD, depthDataH, rows*cols*sizeof(float));
}

void downloadTileDebug(int xRes, int yRes, int zRes, float* voxelsD, float* voxelsH)
{
    const int numBytes = xRes*yRes*zRes*sizeof(float);
    memcpy(voxelsH, voxelsD, numBytes);
}

void projectDebug(float* projMat, float* point, float* projPoint)
{
    projPoint[0] = projMat[0]*point[0] + projMat[1]*point[1] + projMat[2]*point[2] + projMat[3];
    projPoint[1] = projMat[4]*point[0] + projMat[5]*point[1] + projMat[6]*point[2] + projMat[7];
    projPoint[2] = projMat[8]*point[0] + projMat[9]*point[1] + projMat[10]*point[2] + projMat[11];
}

void transformDebug(float* transformMat, float* point, float* transfPoint)
{
    transfPoint[0] = transformMat[0]*point[0] + transformMat[1]*point[1] + transformMat[2]*point[2] + transformMat[3];
    transfPoint[1] = transformMat[4]*point[0] + transformMat[5]*point[1] + transformMat[6]*point[2] + transformMat[7];
    transfPoint[2] = transformMat[8]*point[0] + transformMat[9]*point[1] + transformMat[10]*point[2] + transformMat[11];
}


void integrationKernelWithFixedWeightsDebug(float* voxelD, float* boxToGlobalD,
                                                  int xRes, int yRes, int zRes,
                                                  float xBegin, float yBegin, float zBegin,
                                                  float deltaX, float deltaY, float deltaZ,
                                                  float* depthDataD, int rows, int cols,
                                                  float maxDepth, float minDepth,
                                                  float epsilon, float eta, float delta1, float delta2, float delta3,
                                                  float* projD)
{
    for (int x = 0; x < xRes; ++x) {
        for (int y = 0; y < yRes; ++y) {
            for (int z = 0; z < zRes; ++z) {
                // voxel center in box coordinate frame
                float voxCenterPoint[3];
                voxCenterPoint[0] = xBegin + x*deltaX;
                voxCenterPoint[1] = yBegin + y*deltaY;
                voxCenterPoint[2] = zBegin + z*deltaZ;

                float voxCenterPointG[3];

                transformDebug(boxToGlobalD, voxCenterPoint, voxCenterPointG);
                //std::cout << voxCenterPoint[0] << "; " << voxCenterPoint[1] << "; " << voxCenterPoint[2] << std::endl;
                //std::cout << voxCenterPointG[0] << "; " << voxCenterPointG[1] << "; " << voxCenterPointG[2] << std::endl;

                float voxCenterProj[3];
                projectDebug(projD, voxCenterPointG, voxCenterProj);

                //std::cout << voxCenterProj[0] << "; " << voxCenterProj[1] << "; " << voxCenterProj[2] << std::endl << std::endl;

                if (voxCenterProj[2] >= minDepth && voxCenterProj[2] <= maxDepth)
                {
                    // perspective division
                    int xp,yp;
                    xp = round(voxCenterProj[0]/voxCenterProj[2]);
                    yp = round(voxCenterProj[1]/voxCenterProj[2]);

                    // test if inside image
                    if (xp >= 0 && xp < cols && yp >= 0 && yp < rows) {

                        float depth = depthDataD[yp*cols + xp];

                        if (depth > 0)
                        {
                            float dist = voxCenterProj[2] - depth;

                            if (fabs(dist) > delta1)
                            {
                                const int voxInd = z*xRes*yRes + y*xRes + x;
                                if (fabs(dist) < delta2)
                                {
                                    float weight = ((fabs(dist) - delta1)/(delta2 - delta1))*epsilon;
                                    if (dist < 0)
                                    {
                                        // in front of surface
                                        voxelD[voxInd] -= weight;
                                    }
                                    else if (dist > 0)
                                    {
                                        // behind surface
                                        voxelD[voxInd] += weight;
                                    }

                                }
                                else if(fabs(dist) < delta3)
                                {

                                    if (dist < 0)
                                    {
                                        // in front of surface
                                        voxelD[voxInd] -= epsilon;
                                    }
                                    else if (dist > 0)
                                    {
                                        // behind surface
                                        voxelD[voxInd] += epsilon;
                                    }
                                }
                                else
                                {
                                    if (dist < 0)
                                    {
                                        voxelD[voxInd] -= eta*epsilon;
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}

void integrateDepthMapWithFixedWeights(float* voxelD, float* boxToGlobalD,
                                       int xRes, int yRes, int zRes,
                                       float xBegin, float yBegin, float zBegin,
                                       float deltaX, float deltaY, float deltaZ,
                                       float* depthDataD, int rows, int cols,
                                       float maxDepth, float minDepth,
                                       float epsilon, float eta, float delta1, float delta2, float delta3,
                                       float* projD)
{
    dim3 dimGrid(xRes, yRes);
    dim3 dimBlock(zRes);

    integrationKernelWithFixedWeights<<<dimGrid,dimBlock>>>(voxelD, boxToGlobalD,
                                            xRes, yRes, zRes,
                                            xBegin, yBegin, zBegin,
                                            deltaX, deltaY, deltaZ,
                                            depthDataD, rows, cols,
                                            maxDepth, minDepth,
                                            epsilon, eta, delta1, delta2, delta3,
                                            projD);

    D3D_CUDA_CHECK_ERROR

//    integrationKernelWithFixedWeightsDebug(voxelD, boxToGlobalD,
//                                            xRes, yRes, zRes,
//                                            xBegin, yBegin, zBegin,
//                                            deltaX, deltaY, deltaZ,
//                                            depthDataD, rows, cols,
//                                            maxDepth, minDepth,
//                                            epsilon, eta, delta1, delta2, delta3,
//                                            projD);

}

}
}

